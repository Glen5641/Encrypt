#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

/**
 * Encrypt Program Cuda
 *
 * This program encrypts a file using a degree 2 formula using Cuda
 * Parallelization and then decrypts the file using another degree 2
 * formula.
 *
 * @Author: Clayton Chase Glenn
 */

#define MAX 20
#define DEBUG 0

/** Kernel Function
  * First finds the Thread ID within the block of GPU Threads
  * and if the Thread is Correct, it Encrypts the corresponding
  * Character in the String.
 **/
__global__
void encrypt(char *p, char *c, int a, int b, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid < n) c[tid] = (a*p[tid] + b) % 256;
}

/** Kernel Function
  * First finds the Thread ID within the block of GPU Threads
  * and if the Thread is Correct, it Encrypts the corresponding
  * Character in the String.
 **/
__global__
void decrypt(char *p, char *c, int a, int b, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid < n) c[tid] = (a*p[tid] + b) % 256;
}

/** Kernel Function
  * First finds the Thread ID within the block of GPU Threads
  * and if the Thread is Correct, it checks if the corresponding
  * character in array a matches b.
 **/
__global__
void isMatch(char *p, char *c, int *a, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid < n) {
    if (c[tid] != p[tid]) {
      *a = 1;
    }
  }
}

/**
  * Helper Function
  * Prints an string to standard error showing help
  * for valid arguments in the executable
 **/
void printerror(){
  fprintf(stderr, "Invalid Arguments\n");
  fprintf(stderr, "Correct Form: ./encrypt [File to Encrypt]\n");
  fprintf(stderr, "              or\n");
  fprintf(stderr, "              ./encrypt -n [2^(1:20)]\n");
  exit(0);
}
/**
  * Main Program
  * This Program is for Homework 6 to encrypt some text or show
  * the encryption method of text that is 2 to the power of N
  * characters long all initialized to zero.
 **/
int main(int argc, char **argv) {

  // Declare a buffer of max size to start
  int     N = MAX;
  char *buf;

  // Check for immediate errors in args
  if (argc < 2)                           printerror();
  if (argc == 3 && strcmp(argv[1], "-n")) printerror();

  // If args match for testing, Initiallize the program
  if(!strcmp(argv[1], "-n") && argc == 3){

    // Get total characters from args
    N = strtol(argv[2], NULL, 10);

    // Show error if N isn't within constraints
    if(N < 1 || N > 20) printerror();

    // N <- calc to 2^N as size and allocate space
    N   = (int)pow(2, N);
    buf = (char*)malloc(N*sizeof(char));

    //Initiallize the buffer to Zero
    int i = 0;
    while (i < N) buf[i++] = 48;
  }

  // If 2 args, this means file
  if(argc == 2) {

    // Declare a file pointer, character array, and single character for reading
    FILE *fp;
    char c;
    char chars[1048576];
    int i = 0;

    // Open the file for reading
    fp = fopen(argv[1], "r");

    // If file is null, file does not exist or error
    if (fp == NULL) {
      fprintf(stderr, "Not a Valid File\n");
      return (-1);
    }

    // Read each character and keep within 2^20, add to array
    while((c = fgetc(fp)) != EOF) {
      if (i >= 1048576) {
        fprintf(stderr, "File Too Large\n");
        return (-1);
      }
      chars[i++] = c;
    }

    // Increment i for space and allocate space for buffer
    N = i + 1;
    buf = (char*)malloc(N*sizeof(char));

    // Copy read elements into buffer
    i = 0;
    while(i < N) buf[i] = chars[i++];

    // Close File, not needed anymore
    fclose(fp);
  }

  // Initiallize Character Arrays for Encrypting and manual memset
  char h_p[N];
  char h_c[N];
  char h_r[N];
  int i = 0;
  while (i < N) {
    h_p[i] = buf[i];
    h_c[i] = 32;
    h_r[i++] = 32;
  }

  // Init all other variables
  char *dev_p, *dev_c, *dev_r;
  int *match;
  int h_match = 0;
  int h_a = 171, h_b = 55;
  int r_a = 3,   r_b = 91;
  hipEvent_t start1, start2, start3, startf, stop1, stop2, stop3, stopf;
  hipEventCreate(&start1);
  hipEventCreate(&stop1);
  hipEventCreate(&start2);
  hipEventCreate(&stop2);
  hipEventCreate(&start3);
  hipEventCreate(&stop3);
  hipEventCreate(&startf);
  hipEventCreate(&stopf);
  float final_time1 = 0.0, final_time2 = 0.0, final_time3 = 0.0, final_timef = 0.0;

  // Allocate Memory for match flag
  match = (int*)malloc(sizeof(int));
  *match = 0;

  // Allocate memory in the GPU for the character arrays
  hipMalloc(&dev_p, N*sizeof(char));
  hipMalloc(&dev_c, N*sizeof(char));
  hipMalloc(&dev_r, N*sizeof(char));
  hipMalloc(&match, sizeof(int));

  // Print N for distinguish
  printf("N: %d\n", N);

  // If debug on, show plain text
  if(DEBUG) {
    printf("Plain Text:     ");
    i = 0;
    while(i < N) printf("%c", h_p[i++]);
    printf("\n");
  }

  // Copy the Memory from the arrays to the array pointers
  hipMemcpy(dev_p, h_p, N*sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(dev_c, h_c, N*sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(dev_r, h_r, N*sizeof(char), hipMemcpyHostToDevice);

  // Start Total Time Record
  hipEventRecord(startf);

  // Encrypt the Plain Text and Record Start and Finish
  hipEventRecord(start1);
  encrypt<<<128, 128>>>(dev_p, dev_c, h_a, h_b, N);
  hipEventRecord(stop1);

  // Copy the results from GPU to the CPU
  hipMemcpy(h_c, dev_c, N*sizeof(char), hipMemcpyDeviceToHost);

  // If debug on, show encrypted text
  if(DEBUG) {
    printf("Encrypted Text: ");
    i = 0;
    while(i < N) printf("%c", h_c[i++]);
    printf("\n");
  }

  // Syncronize all blocks and threads in GPU and get time
  hipEventSynchronize(stop1);
  hipEventElapsedTime(&final_time1, start1, stop1);

  // Decrypt the Encrypted Text
  hipEventRecord(stop2);
  decrypt<<<128, 128>>>(dev_c, dev_r, r_a, r_b, N);
  hipEventRecord(stop2);

  // Copy the results from GPU to CPU
  hipMemcpy(h_r, dev_r, N*sizeof(char), hipMemcpyDeviceToHost);

  // If debug on, show decrypted text
  if(DEBUG) {
    printf("Decrypted Text: ", h_r);
    i = 0;
    while(i < N) printf("%c", h_r[i++]);
    printf("\n");
  }

  // Syncronize all blocks and threads in GPU and get time
  hipEventSynchronize(stop2);
  hipEventElapsedTime(&final_time2, start2, stop2);

  // Check if Plain Text and Encrypt<-->Decrypt Text is matching by GPU
  hipEventRecord(start3);
  isMatch<<<128, 128>>>(dev_r, dev_p, match, N);
  hipEventRecord(stop3);

  // Copy the Match Result from GPU to CPU
  hipMemcpy(&h_match, match, sizeof(int), hipMemcpyDeviceToHost);

  // If match is zero, success, else, no success
  if (h_match) fprintf(stdout, "Does not Match\n");
  else         fprintf(stdout, "Does Match\n");

  // Syncronize all blocks and threads in GPU and get time
  hipEventSynchronize(stop3);
  hipEventElapsedTime(&final_time3, start3, stop3);

  // Syncronize all blocks and threads in GPU and get time
  hipEventRecord(stopf);
  hipEventSynchronize(stopf);
  hipEventElapsedTime(&final_timef, startf, stopf);

  // Print Times
  printf("Encrypt Time:   %4.10f seconds\n",   final_time1/1000);
  printf("Decrypt Time:   %4.10f seconds\n",   final_time2/1000);
  printf("Match Time:     %4.10f seconds\n",   final_time3/1000);
  printf("Total Time:     %4.10f seconds\n\n", final_timef/1000);

  // Free the GPU memory
  hipFree(dev_p);
  hipFree(dev_c);
  hipFree(dev_r);
}
